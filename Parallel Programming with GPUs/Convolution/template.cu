#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <wb.h>

#define MASK_WIDTH 5
#define O_TILE_WIDTH 16
#define clamp(x) (min(max((x), 0.0), 1.0))
#define TILE_SIZE O_TILE_WIDTH * O_TILE_WIDTH * 3
#define BLOCK_WIDTH O_TILE_WIDTH + (MASK_WIDTH - 1)

//@@ INSERT CODE HERE 
//implement the tiled 2D convolution kernel with adjustments for channels
//use shared memory to reduce the number of global accesses, handle the boundary conditions when loading input list elements into the shared memory
//clamp your output values

__global__ void convolution_2D(const float* __restrict__ mask, float* P, float* input, int height, int width, int channels) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int K = threadIdx.z;
	int maskRadius = MASK_WIDTH / 2;

	int col = blockIdx.x * blockDim.x + tx;
	int row = blockIdx.y * blockDim.y + ty;

	int row_i = row - ((MASK_WIDTH - 1) * blockIdx.y) - maskRadius;
	int col_i = col - ((MASK_WIDTH - 1) * blockIdx.x) - maskRadius;

	int i, j, k, ind;
	float imagePixel, maskV;
	__shared__ float I[TILE_SIZE];

	k = (ty * blockDim.y + tx) * channels + K;
	ind = (row_i * width + col_i) * channels + K;
	if (row_i >= 0 && row_i < height && col_i >= 0 && col_i < width) {
		I[k] = input[ind];
	}
	else {
		I[k] = 0.0f;
	}

	__syncthreads();

    if(row_i >= 0 && row_i < height && col_i >= 0 && col_i < width) {
        if(tx >= maskRadius && tx < (blockDim.x - maskRadius) && ty >= maskRadius && ty < (blockDim.y - maskRadius)) {
            float accum = 0.0f;
            for(i = 0; i < MASK_WIDTH; i++) {
                for(j = 0; j < MASK_WIDTH; j++) {
                    int xOffset = ty - maskRadius + i;
                    int yOffset = tx - maskRadius + j;
                    imagePixel = I[(xOffset * blockDim.y + yOffset) * channels + K];
                    maskV = mask[i * MASK_WIDTH + j];
                    accum += imagePixel * maskV;
                }
            }
            P[(row_i * width + col_i) * channels + K] = clamp(accum, 0, 1);
        }
    }
}

int main(int argc, char *argv[]) {
  wbArg_t arg;
  int maskRows;
  int maskColumns;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  char *inputMaskFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *hostMaskData;
  float *deviceInputImageData;
  float *deviceOutputImageData;
  float *deviceMaskData;

  arg = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(arg, 0);
  inputMaskFile  = wbArg_getInputFile(arg, 1);

  inputImage   = wbImport(inputImageFile);
  hostMaskData = (float *)wbImport(inputMaskFile, &maskRows, &maskColumns);

  assert(maskRows == MASK_WIDTH);    /* mask height is fixed to 5 */
  assert(maskColumns == MASK_WIDTH); /* mask width is fixed to 5 */

  imageWidth    = wbImage_getWidth(inputImage);
  imageHeight   = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);

  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ INSERT CODE HERE
  //allocate device memory

  int Size = imageWidth * imageHeight * imageChannels * sizeof(float);
  int MSize = maskRows * maskColumns * sizeof(float);

  hipMalloc((void**)&deviceInputImageData, Size);
  hipMalloc((void**)&deviceOutputImageData, Size);
  hipMalloc((void**)&deviceMaskData, MSize);

  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ INSERT CODE HERE
  //copy host memory to device

  hipMemcpy(deviceInputImageData, hostInputImageData, Size, hipMemcpyHostToDevice);
  hipMemcpy(deviceOutputImageData, hostOutputImageData, Size, hipMemcpyHostToDevice);
  hipMemcpy(deviceMaskData, hostMaskData, MSize, hipMemcpyHostToDevice);

  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  //initialize thread block and kernel grid dimensions
  //invoke CUDA kernel	

  dim3 dimBlock(O_TILE_WIDTH, O_TILE_WIDTH, imageChannels);
  dim3 dimGrid((imageWidth - 1) / (O_TILE_WIDTH - MASK_WIDTH) + 1, (imageHeight - 1) / (O_TILE_WIDTH - MASK_WIDTH) + 1, 1);
  convolution_2D <<<dimGrid, dimBlock>>>(deviceMaskData, deviceOutputImageData, deviceInputImageData, imageHeight, imageWidth, imageChannels);

  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ INSERT CODE HERE
  //copy results from device to host

  hipMemcpy(hostOutputImageData, deviceOutputImageData, Size, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(arg, outputImage);

  //@@ INSERT CODE HERE
  //deallocate device memory

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);
  hipFree(deviceMaskData);

  free(hostMaskData);
  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
